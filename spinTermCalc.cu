#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

template <int I>
struct Int2Type
{
  enum { value = I };
};

class Managed {

public:

	size_t size;

	void *operator new(size_t len) {

		void *ptr;

		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();

		return ptr;
	}

	void operator delete(void *ptr) {

		hipDeviceSynchronize();

		hipFree(ptr);
	}

	void sync() {
		hipDeviceSynchronize();
	}
};

enum Spin { SPIN0, SPIN1, SPIN2, SPIN3, SPIN4, SPIN5 };
enum SpinFactor { ZEMACH, COVARIANT, LEGENDRE };

struct SpinTermParams
{
public:

	std::vector<float> * p;
	std::vector<float> * q;
	std::vector<float> * erm;
	std::vector<float> * cosHel;
	std::vector<float> * leg;
	std::vector<float> * spinTerms;

	// Deal with these guys later...
	static const int spin = 2;
	static const int spinType = 0;

	SpinTermParams(int s)
	{
		p = new std::vector<float>(s);
		q = new std::vector<float>(s);
		erm = new std::vector<float>(s);
		cosHel = new std::vector<float>(s);
		leg = new std::vector<float>(s);
		spinTerms = new std::vector<float>(s);
	}

	~SpinTermParams()
	{
		delete p;
		delete q;
		delete erm;
		delete cosHel;
		delete leg;
		delete spinTerms;
	}

};

struct KernelParams
{
	float * p;
	float * q;
	float * erm;
	float * cosHel;
	float * leg;
	float * spinTerms;

	KernelParams(int s)
	{
		p = new float[s];
		q = new float[s];
		erm = new float[s];
		cosHel = new float[s];
		leg = new float[s];
		spinTerms = new float[s];
	}

	~KernelParams()
	{
		delete p;
		delete q;
		delete erm;
		delete cosHel;
		delete leg;
		delete spinTerms;
	}

};

// struct KernelParamsL
// {
// 	float * cosHel;
// 	float * leg;
// };

class FloatArr : public Managed
{

public:

	int size;
	float * data;

	FloatArr() : size(0), data(0)
	{

	}

	FloatArr(std::vector<float> * a) : size(a->size())
	{
		// Allocate unified memory
		realloc_(a->size());

		// Copy C array from vector
		memcpy(data, a->data(), a->size() * sizeof(float));
	}

	FloatArr(const FloatArr & a) : size(a.size)
	{
		realloc_(a.size);
		memcpy(data, a.data, a.size * sizeof(float));
	}

	~FloatArr() { hipFree(data); }

	FloatArr& operator=(std::vector<float> * a)
	{
		size = a->size();
		realloc_(a->size());
		memcpy(data, a->data(), size * sizeof(float));
		return *this;
    }

	__host__ __device__
    float& operator[](int pos) { return data[pos]; }

private:

	void realloc_(int s)
	{
		// hipFree(data);
	    hipMallocManaged(&data, s * sizeof(float));
		hipDeviceSynchronize();
	}

};

class KernelParamsL : public Managed
{

public:

	FloatArr cosHel;
	FloatArr leg;

	KernelParamsL() {}

	KernelParamsL(FloatArr cosHel_, FloatArr leg_) : cosHel(cosHel_), leg(leg_) {}

	void prefetch()
	{
		// Would prefer a loop over elements

		int device = -1;
		hipGetDevice(&device);

		hipMemPrefetchAsync(&cosHel, cosHel.size * sizeof(float), device, NULL);
		hipMemPrefetchAsync(&leg, leg.size * sizeof(float), device, NULL);
		hipDeviceSynchronize();
	}

};

// Spin functions

template<typename Spin>
__device__
float legFunc(float cosHel)
{
    return 1.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN0>>(float cosHel)
{
    return 1.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN1>>(float cosHel)
{
    return -2.0 * cosHel;
}

template<>
__device__
float legFunc<Int2Type<SPIN2>>(float cosHel)
{
    return 4.0*(3.0*cosHel*cosHel - 1.0)/3.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN3>>(float cosHel)
{
    return -8.0*(5.0*cosHel*cosHel*cosHel - 3.0*cosHel)/5.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN4>>(float cosHel)
{
    return 16.0*(35.0*cosHel*cosHel*cosHel*cosHel - 30.0*cosHel*cosHel + 3.0)/35.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN5>>(float cosHel)
{
    return -32.0*(63.0*cosHel*cosHel*cosHel*cosHel*cosHel - 70.0*cosHel*cosHel*cosHel + 15.0*cosHel)/63.0;
}

// Cov factors

template<typename Spin>
__device__
float covFactor(float erm)
{
    return 1.0;
}

template<>
__device__
float covFactor<Int2Type<SPIN0>>(float erm)
{
    return 1.0;
}

template<>
__device__
float covFactor<Int2Type<SPIN1>>(float erm)
{
    return erm;
}

template<>
__device__
float covFactor<Int2Type<SPIN2>>(float erm)
{
    return erm*erm + 0.5;
}

template<>
__device__
float covFactor<Int2Type<SPIN3>>(float erm)
{
    return erm*(erm*erm + 1.5);
}

template<>
__device__
float covFactor<Int2Type<SPIN4>>(float erm)
{
    return (8.*erm*erm*erm*erm + 24.*erm*erm + 3.)/35.;
}


template<typename Spin>
__global__
void legKern(const int n, KernelParamsL * params)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        params->leg[i] = legFunc<Spin>(params->cosHel[i]);
    }

}

template<typename Spin>
__global__
void spinTermZemachKern(const int n, KernelParams params)
{

	// Get an instance of our Int2Type type, so that
    // s.value is out integer spin (the enum value)

    Spin s;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){

        params.leg[i] = legFunc<Spin>(params.cosHel[i]);

        float pProd = params.p[i] * params.q[i];

        params.spinTerms[i] = params.leg[i] * pow(pProd, s.value);
    }

}

template<typename Spin>
__global__
void spinTermCovKern(const int n, KernelParams params)
{

	// Get an instance of our Int2Type type, so that
    // s.value is out integer spin (the enum value)

    Spin s;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){

        params.leg[i] = legFunc<Spin>(params.cosHel[i]);

        float pProd = params.p[i] * params.q[i];

        params.spinTerms[i] = params.leg[i] * pow(pProd, s.value) * covFactor<Spin>(params.erm[i]);
    }

}

void calcLegendrePolyManaged(const SpinTermParams & inParams)
{

	int n = inParams.cosHel->size();

    KernelParamsL * kParams = new KernelParamsL(inParams.cosHel, inParams.leg);

	// kParams->prefetch();

	int blockSize = 128;
	int numBlocks = (n + blockSize - 1) / blockSize;

	legKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);

	kParams->sync();

    std::cout << kParams->leg[n - 1] << std::endl;

	inParams.leg->insert(inParams.leg->begin(), &kParams->leg[0], &kParams->leg[n]);

    std::cout << (inParams.leg)->at(n - 1) << std::endl;
}

// void calcLegendrePoly(SpinTermParams inParams)
// {
//
// 	int n = inParams.cosHel->size();
//
// 	// KernelParams * kParams = new KernelParams();
//     KernelParamsL kParams;
//
//     hipError_t mallocStatus;
//
// 	mallocStatus = hipMallocManaged(&(kParams.leg), n * sizeof(float));
//     if (mallocStatus != hipSuccess) std::cout << mallocStatus << std::endl;
//
// 	mallocStatus = hipMallocManaged(&(kParams.cosHel), n * sizeof(float));
//     if (mallocStatus != hipSuccess) std::cout << mallocStatus << std::endl;
//
//     // Now points somewhere different!
// 	// kParams.cosHel = inParams.cosHel->data();
//
//     *kParams.cosHel = *inParams.cosHel->data();
//
// 	int device = -1;
//
// 	hipGetDevice(&device);
//
// 	hipMemPrefetchAsync(kParams.leg, n * sizeof(float), device, NULL);
// 	hipMemPrefetchAsync(kParams.cosHel, n * sizeof(float), device, NULL);
//
// 	int blockSize = 128;
// 	int numBlocks = (n + blockSize - 1) / blockSize;
//
// 	legKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
//
//     exit(0);
//
// 	hipError_t cudaStatus = hipDeviceSynchronize();
//
// 	if (cudaStatus != hipSuccess) {
// 	    std::cout << "sync failed" << std::endl;
// 	}
//
// 	inParams.leg->insert(inParams.leg->end(), &kParams.leg[0], &kParams.leg[n]);
//
//     hipFree(kParams.leg);
//     hipFree(kParams.cosHel);
//
// }
//
// void calcSpinTerm(SpinTermParams inParams)
// {
//
// 	int n = inParams.cosHel->size;
//
// 	bool covariant = inParams.spinType == COVARIANT;
//
//     std::cout << n << " " << covariant << std::endl;
//
// 	KernelParams kParams;
//
// 	hipMallocManaged(&kParams.spinTerms, n * sizeof(float));
// 	hipMallocManaged(&kParams.leg, n * sizeof(float));
// 	hipMallocManaged(&kParams.cosHel, n * sizeof(float));
// 	hipMallocManaged(&kParams.p, n * sizeof(float));
// 	hipMallocManaged(&kParams.q, n * sizeof(float));
// 	if (covariant) hipMallocManaged(&kParams.erm, n * sizeof(float));
//
// 	// Init on device (we can do that as memory is 'unified')
//
// 	kParams.cosHel = inParams.cosHel->data();
// 	kParams.p = inParams.p->data();
// 	kParams.q = inParams.q->data();
// 	if (covariant) kParams.erm = inParams.erm->data();
//
// 	int device = -1;
//
// 	hipGetDevice(&device);
//
// 	hipMemPrefetchAsync(kParams.spinTerms, n * sizeof(float), device, NULL);
// 	hipMemPrefetchAsync(kParams.leg, n * sizeof(float), device, NULL);
// 	hipMemPrefetchAsync(kParams.cosHel, n * sizeof(float), device, NULL);
// 	hipMemPrefetchAsync(kParams.p, n * sizeof(float), device, NULL);
// 	hipMemPrefetchAsync(kParams.q, n * sizeof(float), device, NULL);
// 	if (covariant) hipMemPrefetchAsync(kParams.erm, n * sizeof(float), device, NULL);
//
// 	int blockSize = 128;
// 	int numBlocks = (n + blockSize - 1) / blockSize;
//
// 	// spinTermKern<Int2Type<SPIN1>, Int2Type<ZEMACH>><<<numBlocks, blockSize>>>(n, cosHel, p, q, out, leg);
// 	// spinTermKern<Int2Type<spin>, Int2Type<spinType>><<<numBlocks, blockSize>>>(n, cosHel, p, q, out, leg);
//
// 	if (!covariant) {
// 		spinTermZemachKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
// 	} else {
// 		spinTermCovKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
// 	}
//
// 	hipError_t cudaStatus = hipDeviceSynchronize();
//
// 	if (cudaStatus != hipSuccess) {
// 	    std::cout << "sync failed" << std::endl;
// 	}
//
// 	inParams.leg->insert(inParams.leg->end(), &kParams.leg[0], &kParams.leg[n]);
// 	inParams.spinTerms->insert(inParams.spinTerms->end(), &kParams.spinTerms[0], &kParams.spinTerms[n]);
// }

int main(int argc, char const *argv[]) {

    SpinTermParams pars(int(1E4));

	std::fill(pars.cosHel->begin(), pars.cosHel->end(), 0.2);
	std::fill(pars.q->begin(), pars.q->end(), 1.0);
	std::fill(pars.p->begin(), pars.p->end(), 1.0);
	std::fill(pars.erm->begin(), pars.erm->end(), 1.0);
	std::fill(pars.leg->begin(), pars.leg->end(), 1.0);

	calcLegendrePolyManaged(pars);

	std::cout << (pars.leg)->at(5) << std::endl;

	return 0;
}
