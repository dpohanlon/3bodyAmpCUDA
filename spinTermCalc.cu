#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <cmath>

// From http://www.drdobbs.com/genericprogramming-mappings-between-type/184403750
template <int I>
struct Int2Type
{
  enum { value = I };
};

class Managed {

public:

	size_t size;

	void *operator new(size_t len) {

		void *ptr;

		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();

		return ptr;
	}

	void operator delete(void *ptr) {

		hipDeviceSynchronize();

		hipFree(ptr);
	}

	void sync() {
		hipDeviceSynchronize();
	}
};

enum Spin { SPIN0, SPIN1, SPIN2, SPIN3, SPIN4, SPIN5 };
enum SpinFactor { ZEMACH, COVARIANT, LEGENDRE };

struct SpinTermParams
{
public:

	std::vector<float> * p;
	std::vector<float> * q;
	std::vector<float> * erm;
	std::vector<float> * cosHel;
	std::vector<float> * leg;
	std::vector<float> * spinTerms;

	// Deal with these guys later...
	static const int spin = 4;
	static const int spinType = 1;

	SpinTermParams(int s)
	{
		p = new std::vector<float>(s);
		q = new std::vector<float>(s);
		erm = new std::vector<float>(s);
		cosHel = new std::vector<float>(s);
		leg = new std::vector<float>(s);
		spinTerms = new std::vector<float>(s);
	}

	~SpinTermParams()
	{
		delete p;
		delete q;
		delete erm;
		delete cosHel;
		delete leg;
		delete spinTerms;
	}

};

struct ResParams
{
public:

    float resMass;
    float resWidth;

    std::vector<float> * mass;
	std::vector<float> * qTerm;
    std::vector<float> * ffRatioP;
    std::vector<float> * ffRatioR;
	std::vector<float> * spinTerms;

    std::vector<float> * ampRe;
    std::vector<float> * ampIm;

	// Deal with these guys later...
	static const int spin = 4;
	static const int spinType = 1;

	ResParams(int s)
	{
		mass = new std::vector<float>(s);
		qTerm = new std::vector<float>(s);
		ffRatioP = new std::vector<float>(s);
		ffRatioR = new std::vector<float>(s);
		spinTerms = new std::vector<float>(s);

        ampRe = new std::vector<float>(s);
        ampIm = new std::vector<float>(s);
	}

	~ResParams()
	{
        delete mass;
		delete qTerm;
		delete ffRatioP;
		delete ffRatioR;
		delete spinTerms;

        delete ampRe;
        delete ampIm;
	}

};

class FloatArr : public Managed
{

public:

	int size;
	float * data;

	FloatArr() : size(0), data(0)
	{

	}

	FloatArr(std::vector<float> * a) : size(a->size())
	{
		// Allocate unified memory
		realloc_(a->size());

		// Copy C array from vector
		memcpy(data, a->data(), a->size() * sizeof(float));
	}

	FloatArr(const FloatArr & a) : size(a.size)
	{
		realloc_(a.size);
		memcpy(data, a.data, a.size * sizeof(float));
	}

	~FloatArr() { hipFree(data); }

	FloatArr& operator=(std::vector<float> * a)
	{
		size = a->size();
		realloc_(a->size());
		memcpy(data, a->data(), size * sizeof(float));
		return *this;
    }

	void prefetch()
	{
		int device = -1;
		hipGetDevice(&device);

		hipMemPrefetchAsync(data, size * sizeof(float), device, NULL);
		hipMemPrefetchAsync(&size, sizeof(int), device, NULL);
	}

	__host__ __device__
    float& operator[](int pos) { return data[pos]; }

private:

	void realloc_(int s)
	{
		// hipFree(data);
	    hipMallocManaged(&data, s * sizeof(float));
		hipDeviceSynchronize();
	}

};

class KernelParamsL : public Managed
{

public:

	FloatArr cosHel;
	FloatArr leg;

	KernelParamsL() {}

	KernelParamsL(FloatArr cosHel_, FloatArr leg_) : cosHel(cosHel_), leg(leg_) {}

	void prefetch()
	{
		cosHel.prefetch();
		leg.prefetch();
	}

};

class KernelParams : public Managed
{

public:

	FloatArr cosHel;
	FloatArr leg;
	FloatArr p;
	FloatArr q;
	FloatArr erm;
	FloatArr spinTerms;

	KernelParams() {}

	void prefetch()
	{
		cosHel.prefetch();
		leg.prefetch();
		p.prefetch();
		q.prefetch();
		spinTerms.prefetch();
        if (erm.size > 0) erm.prefetch();
	}

};

class KernelResParams : public Managed
{

public:

    // float resMass;
    // float resWidth;

    FloatArr mass;
    FloatArr qTerm;
    FloatArr ffRatioP;
    FloatArr ffRatioR;
    FloatArr spinTerms;
    FloatArr ampRe;
    FloatArr ampIm;

	KernelResParams() {}

	void prefetch()
	{
        mass.prefetch();
    	qTerm.prefetch();
        ffRatioP.prefetch();
        ffRatioR.prefetch();
    	spinTerms.prefetch();
        ampRe.prefetch();
        ampIm.prefetch();
	}

};

// Spin functions

template<typename Spin>
__host__ __device__
float legFunc(float cosHel)
{
    return 1.0;
}

template<>
__host__ __device__
float legFunc<Int2Type<SPIN0>>(float cosHel)
{
    return 1.0;
}

template<>
__host__ __device__
float legFunc<Int2Type<SPIN1>>(float cosHel)
{
    return -2.0 * cosHel;
}

template<>
__host__ __device__
float legFunc<Int2Type<SPIN2>>(float cosHel)
{
    return 4.0*(3.0*cosHel*cosHel - 1.0)/3.0;
}

template<>
__host__ __device__
float legFunc<Int2Type<SPIN3>>(float cosHel)
{
    return -8.0*(5.0*cosHel*cosHel*cosHel - 3.0*cosHel)/5.0;
}

template<>
__host__ __device__
float legFunc<Int2Type<SPIN4>>(float cosHel)
{
    return 16.0*(35.0*cosHel*cosHel*cosHel*cosHel - 30.0*cosHel*cosHel + 3.0)/35.0;
}

template<>
__host__ __device__
float legFunc<Int2Type<SPIN5>>(float cosHel)
{
    return -32.0*(63.0*cosHel*cosHel*cosHel*cosHel*cosHel - 70.0*cosHel*cosHel*cosHel + 15.0*cosHel)/63.0;
}

// For branching

float legFunc0(float cosHel)
{
    return 1.0;
}

float legFunc1(float cosHel)
{
    return -2.0 * cosHel;
}

float legFunc2(float cosHel)
{
    return 4.0*(3.0*cosHel*cosHel - 1.0)/3.0;
}

float legFunc3(float cosHel)
{
    return -8.0*(5.0*cosHel*cosHel*cosHel - 3.0*cosHel)/5.0;
}

// Cov factors

template<typename Spin>
__host__ __device__
float covFactor(float erm)
{
    return 1.0;
}

template<>
__host__ __device__
float covFactor<Int2Type<SPIN0>>(float erm)
{
    return 1.0;
}

template<>
__host__ __device__
float covFactor<Int2Type<SPIN1>>(float erm)
{
    return erm;
}

template<>
__host__ __device__
float covFactor<Int2Type<SPIN2>>(float erm)
{
    return erm*erm + 0.5;
}

template<>
__host__ __device__
float covFactor<Int2Type<SPIN3>>(float erm)
{
    return erm*(erm*erm + 1.5);
}

template<>
__host__ __device__
float covFactor<Int2Type<SPIN4>>(float erm)
{
    return (8.*erm*erm*erm*erm + 24.*erm*erm + 3.)/35.;
}

__global__
void ampKern(const int n, const float resMass, const float resWidth, KernelResParams * params)
{
	// All threads handle blockDim.x * gridDim.x
	// consecutive elements (interleaved partitioning)

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        float totWidth = resWidth * params->qTerm[i];
        totWidth *= (resMass / params->mass[i]);
        totWidth *= params->ffRatioP[i] * params->ffRatioR[i];

        float m2 = params->mass[i] * params->mass[i];
        float m2Term = resMass * resMass - m2;

        float scale = params->spinTerms[i];
        scale /= m2Term * m2Term + resMass * resMass * totWidth * totWidth;
        scale *= params->ffRatioP[i] * params->ffRatioR[i]; // Optional -> template specialise?

        params->ampRe[i] = m2Term * scale;
        params->ampIm[i] = resMass * totWidth * scale;
    }

}

template<typename Spin>
__global__
void legKern(const int n, KernelParamsL * params)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        params->leg[i] = legFunc<Spin>(params->cosHel[i]);
    }

}

__global__
void legKernBranch(const int n, const int spin, KernelParamsL * params)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if (spin == 0) params->leg[i] = legFunc0(params->cosHel[i]);
        if (spin == 1) params->leg[i] = legFunc1(params->cosHel[i]);
        if (spin == 2) params->leg[i] = legFunc2(params->cosHel[i]);
        if (spin == 3) params->leg[i] = legFunc3(params->cosHel[i]);
    }

}

template<typename Spin>
__global__
void spinTermZemachKern(const int n, KernelParams * params)
{

	// Get an instance of our Int2Type type, so that
    // s.value is out integer spin (the enum value)

    Spin s;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){

        params->leg[i] = legFunc<Spin>(params->cosHel[i]);

        float pProd = params->p[i] * params->q[i];

        params->spinTerms[i] = params->leg[i] * pow(pProd, s.value);
    }

}

template<typename Spin>
__global__
void spinTermCovKern(const int n, KernelParams * params)
{
	// Get an instance of our Int2Type type, so that
    // s.value is out integer spin (the enum value)

    Spin s;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){

        params->leg[i] = legFunc<Spin>(params->cosHel[i]);

        float pProd = params->p[i] * params->q[i];

        params->spinTerms[i] = params->leg[i] * pow(pProd, s.value) * covFactor<Spin>(params->erm[i]);
    }

}

void calcLegendrePolyManaged(const SpinTermParams & inParams)
{
	int n = inParams.cosHel->size();

	KernelParamsL * kParams = new KernelParamsL();

	kParams->cosHel = inParams.cosHel;
	kParams->leg = inParams.leg;

	kParams->prefetch();

	int blockSize = 128;
	int numBlocks = (n + blockSize - 1) / blockSize;

	legKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);

	kParams->sync();

	inParams.leg->insert(inParams.leg->begin(), &kParams->leg[0], &kParams->leg[n]);

	delete kParams;
}

void calcSpinTerm(const SpinTermParams & inParams)
{
	int n = inParams.cosHel->size();

	bool covariant = inParams.spinType == COVARIANT;

    std::cout << covariant << std::endl;

	KernelParams * kParams = new KernelParams();

	kParams->cosHel = inParams.cosHel;
	kParams->leg = inParams.leg;
	kParams->p = inParams.p;
	kParams->q = inParams.q;
	kParams->spinTerms = inParams.spinTerms;
	if (covariant) kParams->erm = inParams.erm;

	kParams->prefetch();

	int blockSize = 128;
	int numBlocks = (n + blockSize - 1) / blockSize;

	if (!covariant) {
		spinTermZemachKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
	} else {
		spinTermCovKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
	}

	kParams->sync();

	inParams.leg->insert(inParams.leg->begin(), &kParams->leg[0], &kParams->leg[n]);
	inParams.spinTerms->insert(inParams.spinTerms->begin(), &kParams->spinTerms[0], &kParams->spinTerms[n]);

	delete kParams;
}

void calcAmp(const ResParams & inParams)
{
	int n = inParams.mass->size();

	KernelResParams * kParams = new KernelResParams();

	kParams->spinTerms = inParams.spinTerms;
    kParams->mass = inParams.mass;
	kParams->qTerm = inParams.qTerm;
    kParams->ffRatioP = inParams.ffRatioP;
    kParams->ffRatioR = inParams.ffRatioR;

    kParams->ampRe = inParams.ampRe;
    kParams->ampIm = inParams.ampIm;

    // kParams->resMass = inParams.resMass;
    // kParams->resWidth = inParams.resWidth;

	kParams->prefetch();

	int blockSize = 128;
	int numBlocks = (n + blockSize - 1) / blockSize;

    for (int i = 0; i < 100; i++){
        ampKern<<<numBlocks, blockSize>>>(n, inParams.resMass, inParams.resWidth, kParams);
    }

	kParams->sync();

	inParams.ampRe->insert(inParams.ampRe->begin(), &kParams->ampRe[0], &kParams->ampRe[n]);
    inParams.ampIm->insert(inParams.ampIm->begin(), &kParams->ampIm[0], &kParams->ampIm[n]);

	delete kParams;
}

void calcSpinTermCPU(const SpinTermParams & inParams)
{

    for (int i = 0; i < inParams.cosHel->size(); i++) {
        inParams.leg->at(i) = legFunc<Int2Type<inParams.spin>>(inParams.cosHel->at(i));
		float pProd = inParams.p->at(i) * inParams.q->at(i);
		inParams.spinTerms->at(i) = inParams.leg->at(i) * pow(pProd, inParams.spin) * covFactor<Int2Type<inParams.spin>>(inParams.erm->at(i));
    }
}

void calcAmpCPU(const ResParams & inParams)
{
    float resMass = inParams.resMass;
    float resWidth = inParams.resWidth;

    for (int i = 0; i < inParams.mass->size(); i++) {
        float totWidth = resWidth * inParams.qTerm->at(i);
        totWidth *= (resMass / inParams.mass->at(i));
        totWidth *= inParams.ffRatioP->at(i) * inParams.ffRatioR->at(i);

        float m2 = inParams.mass->at(i) * inParams.mass->at(i);
        float m2Term = resMass * resMass - m2;

        float scale = inParams.spinTerms->at(i);
        scale /= m2Term * m2Term + resMass * resMass * totWidth * totWidth;
        scale *= inParams.ffRatioP->at(i) * inParams.ffRatioR->at(i); // Optional -> template specialise?

        inParams.ampRe->at(i) = m2Term * scale;
        inParams.ampIm->at(i) = resMass * totWidth * scale;
    }
}

int main(int argc, char const *argv[]) {

    // SpinTermParams pars(int(1E8));

	// Things to consider:
	// floats -> keep an eye on the precision
	// Might want to keep some things in GPU memory (e.g., spin terms) for further calculations

	// std::fill(pars.cosHel->begin(), pars.cosHel->end(), 0.2);
	// std::fill(pars.q->begin(), pars.q->end(), 0.05);
	// std::fill(pars.p->begin(), pars.p->end(), 0.003);
	// std::fill(pars.erm->begin(), pars.erm->end(), 3.3);
	// std::fill(pars.leg->begin(), pars.leg->end(), 1.0);
	// std::fill(pars.spinTerms->begin(), pars.spinTerms->end(), 1.0);
    //
	// calcSpinTerm(pars);
	// // calcSpinTermCPU(pars);
    //
	// std::cout << (pars.leg)->at(5) << std::endl;
	// std::cout << (pars.spinTerms)->at(5) << std::endl;

    ResParams parsR(int(1E8));

	std::fill(parsR.qTerm->begin(), parsR.qTerm->end(), 0.05);
	std::fill(parsR.mass->begin(), parsR.mass->end(), 0.3);
	std::fill(parsR.ffRatioP->begin(), parsR.ffRatioP->end(), 3.3);
	std::fill(parsR.ffRatioR->begin(), parsR.ffRatioR->end(), 1.0);
	std::fill(parsR.spinTerms->begin(), parsR.spinTerms->end(), 1.0);

	std::fill(parsR.ampRe->begin(), parsR.ampRe->end(), 1.0);
	std::fill(parsR.ampIm->begin(), parsR.ampIm->end(), 1.0);

    parsR.resMass = 1.0;
    parsR.resWidth = 0.1;

	calcAmp(parsR);

    // for (int i = 0; i < 100; i++){
    // 	calcAmpCPU(parsR);
    // }

    std::cout << (parsR.ampRe)->at(5) << std::endl;
	std::cout << (parsR.ampIm)->at(5) << std::endl;

	return 0;
}
