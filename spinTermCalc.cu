#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

template <int I>
struct Int2Type
{
  enum { value = I };
};

enum Spin { SPIN0, SPIN1, SPIN2, SPIN3, SPIN4, SPIN5 };
enum SpinFactor { ZEMACH, COVARIANT, LEGENDRE };

struct SpinTermParams
{
	std::vector<float> * p;
	std::vector<float> * q;
	std::vector<float> * erm;
	std::vector<float> * cosHel;
	std::vector<float> * leg;
	std::vector<float> * spinTerms;

	// Deal with these guys later...
	static const int spin = 0;
	static const int spinType = 0;
};

struct KernelParams
{
	float * p;
	float * q;
	float * erm;
	float * cosHel;
	float * leg;
	float * spinTerms;
};

// Spin functions

template<typename Spin>
__device__
float legFunc(float cosHel)
{
    return 1.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN0>>(float cosHel)
{
    return 1.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN1>>(float cosHel)
{
    return -2.0 * cosHel;
}

template<>
__device__
float legFunc<Int2Type<SPIN2>>(float cosHel)
{
    return 4.0*(3.0*cosHel*cosHel - 1.0)/3.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN3>>(float cosHel)
{
    return -8.0*(5.0*cosHel*cosHel*cosHel - 3.0*cosHel)/5.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN4>>(float cosHel)
{
    return 16.0*(35.0*cosHel*cosHel*cosHel*cosHel - 30.0*cosHel*cosHel + 3.0)/35.0;
}

template<>
__device__
float legFunc<Int2Type<SPIN5>>(float cosHel)
{
    return -32.0*(63.0*cosHel*cosHel*cosHel*cosHel*cosHel - 70.0*cosHel*cosHel*cosHel + 15.0*cosHel)/63.0;
}

// Cov factors

template<typename Spin>
__device__
float covFactor(float erm)
{
    return 1.0;
}

template<>
__device__
float covFactor<Int2Type<SPIN0>>(float erm)
{
    return 1.0;
}

template<>
__device__
float covFactor<Int2Type<SPIN1>>(float erm)
{
    return erm;
}

template<>
__device__
float covFactor<Int2Type<SPIN2>>(float erm)
{
    return erm*erm + 0.5;
}

template<>
__device__
float covFactor<Int2Type<SPIN3>>(float erm)
{
    return erm*(erm*erm + 1.5);
}

template<>
__device__
float covFactor<Int2Type<SPIN4>>(float erm)
{
    return (8.*erm*erm*erm*erm + 24.*erm*erm + 3.)/35.;
}


template<typename Spin>
__global__
void legKern(const int n, KernelParams params)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
    	params.leg[i] = legFunc<Spin>(params.cosHel[i]);
    }

}

template<typename Spin>
__global__
void spinTermZemachKern(const int n, KernelParams params)
{

	// Get an instance of our Int2Type type, so that
    // s.value is out integer spin (the enum value)

    Spin s;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){

    	params.leg[i] = legFunc<Spin>(params.cosHel[i]);

		float pProd = params.p[i] * params.q[i];

		params.spinTerms[i] = params.leg[i] * pow(pProd, s.value);
    }

}

template<typename Spin>
__global__
void spinTermCovKern(const int n, KernelParams params)
{

	// Get an instance of our Int2Type type, so that
    // s.value is out integer spin (the enum value)

    Spin s;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){

    	params.leg[i] = legFunc<Spin>(params.cosHel[i]);

		float pProd = params.p[i] * params.q[i];

		params.spinTerms[i] = params.leg[i] * pow(pProd, s.value) * covFactor<Spin>(params.erm[i]);
    }

}

void calcLegendrePoly(SpinTermParams inParams)
{

	int n = inParams.cosHel->size();

	KernelParams kParams;

	hipMallocManaged(&kParams.leg, n * sizeof(float));
    hipMallocManaged(&kParams.cosHel, n * sizeof(float));

	kParams.cosHel = inParams.cosHel->data();

	int device = -1;

    hipGetDevice(&device);

	hipMemPrefetchAsync(kParams.leg, n * sizeof(float), device, NULL);
	hipMemPrefetchAsync(kParams.cosHel, n * sizeof(float), device, NULL);

	int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / blockSize;

    legKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);

    hipError_t cudaStatus = hipDeviceSynchronize();

    if (cudaStatus != hipSuccess) {
        std::cout << "sync failed" << std::endl;
    }

	inParams.leg->insert(inParams.leg->end(), &kParams.leg[0], &kParams.leg[n]);

}

void calcSpinTerm(SpinTermParams inParams)
{

	int n = inParams.cosHel->size();

	bool covariant = inParams.spinType == COVARIANT;

	KernelParams kParams;

    hipMallocManaged(&kParams.spinTerms, n * sizeof(float));
	hipMallocManaged(&kParams.leg, n * sizeof(float));
    hipMallocManaged(&kParams.cosHel, n * sizeof(float));
	hipMallocManaged(&kParams.p, n * sizeof(float));
	hipMallocManaged(&kParams.q, n * sizeof(float));
	if (covariant) hipMallocManaged(&kParams.erm, n * sizeof(float));

	// Init on device (we can do that as memory is 'unified')

	kParams.cosHel = inParams.cosHel->data();
	kParams.p = inParams.p->data();
	kParams.q = inParams.q->data();
	if (covariant) kParams.erm = inParams.erm->data();

	int device = -1;

    hipGetDevice(&device);

    hipMemPrefetchAsync(kParams.spinTerms, n * sizeof(float), device, NULL);
	hipMemPrefetchAsync(kParams.leg, n * sizeof(float), device, NULL);
	hipMemPrefetchAsync(kParams.cosHel, n * sizeof(float), device, NULL);
	hipMemPrefetchAsync(kParams.p, n * sizeof(float), device, NULL);
    hipMemPrefetchAsync(kParams.q, n * sizeof(float), device, NULL);
	if (covariant) hipMemPrefetchAsync(kParams.erm, n * sizeof(float), device, NULL);

	int blockSize = 128;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // spinTermKern<Int2Type<SPIN1>, Int2Type<ZEMACH>><<<numBlocks, blockSize>>>(n, cosHel, p, q, out, leg);
	// spinTermKern<Int2Type<spin>, Int2Type<spinType>><<<numBlocks, blockSize>>>(n, cosHel, p, q, out, leg);

	if (!covariant) {
		spinTermZemachKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
	} else {
		spinTermCovKern<Int2Type<inParams.spin>><<<numBlocks, blockSize>>>(n, kParams);
	}

    hipError_t cudaStatus = hipDeviceSynchronize();

    if (cudaStatus != hipSuccess) {
        std::cout << "sync failed" << std::endl;
    }

	inParams.leg->insert(inParams.leg->end(), &kParams.leg[0], &kParams.leg[n]);
	inParams.spinTerms->insert(inParams.spinTerms->end(), &kParams.spinTerms[0], &kParams.spinTerms[n]);
}

int main(int argc, char const *argv[]) {

	return 0;
}
